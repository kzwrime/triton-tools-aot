

extern "C" {
#include "fused_moe_kernel_fp8.h"
// #include "kernels/fused_moe_kernel_fp8/fused_moe_kernel_fp8.0860b04c2b25c33e4b2ae8317ea6cbd56d36aaeee2a3cc9d233fb26749410eb1_0d1d2d3d4d5d6d7d8d9d10d11d1213d14c15d16c17d18d19c20d21c22d23d24c.h"
}
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <typeinfo>


#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char *err_str; \
            hipDrvGetErrorString(err, &err_str); \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << err_str << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main(){
    int n = 100;
    // Initialize CUDA
    std::cout << "Initializing CUDA..." << std::endl;
    CUDA_CHECK(hipInit(0));
    hipDevice_t device;
    CUDA_CHECK(hipDeviceGet(&device, 0));
    hipCtx_t context;
    CUDA_CHECK(hipCtxCreate(&context, 0, device));

    // hipDeviceptr_t d_vecA, d_vecB, d_vecC;
    // CUDA_CHECK(hipMalloc(&d_vecA, n * sizeof(float)));
    load_fused_moe_kernel_fp8();
    // unload_fused_moe_kernel_fp8();
    // load_fused_moe_kernel_fp8_0860b04c2b25c33e4b2ae8317ea6cbd56d36aaeee2a3cc9d233fb26749410eb1_0d1d2d3d4d5d6d7d8d9d10d11d1213d14c15d16c17d18d19c20d21c22d23d24c();

    hipError_t result = fused_moe_kernel_fp8_constexpr_dispatcher_with_grid(
        hipStreamPerThread,
        0x7ff388641e00,             // a_ptr = 
        0x7ff31e000000,             // b_ptr = 
        0x7ff388600000,             // c_ptr = 
        0x7ff388642200,             // a_scale_ptr = 
        0x7ff388631000,             // b_scale_ptr = 
        0x7ff38863fe00,             // topk_weights_ptr = 
        0x7ff388681800,             // sorted_token_ids_ptr = 
        0x7ff388641800,             // expert_ids_ptr = 
        0x7ff388641c00,             // num_tokens_post_padded_ptr = 
        7168,               // N = 
        128,                // K = 
        448,                // EM = 
        7,              // num_valid_tokens = 
        128,                // stride_am = 
        1,              // stride_ak = 
        917504,             // stride_be = 
        1,              // stride_bk = 
        128,                // stride_bn = 
        7168,               // stride_cm = 
        1,              // stride_cn = 
        1,              // stride_asm = 
        1,              // stride_ask = 
        56,             // stride_bse = 
        1,              // stride_bsk = 
        1,              // stride_bsn = 
        128,                // group_n = 
        128,                // group_k = 
        64,             // BLOCK_SIZE_M = 
        64,             // BLOCK_SIZE_N = 
        128,                // BLOCK_SIZE_K = 
        32,             // GROUP_SIZE_M = 
        1,              // MUL_ROUTED_WEIGHT = 
        1,              // top_k = 
        1,              // use_fp8_w8a8 = 
        0,              // use_int8_w8a16 = 
        4,              // num_warps = 
        3,              // num_stages = 
        784,                // gridx = 
        1,              // gridy = 
        1              // gridz = 
    );
    CUDA_CHECK(result);
}
