#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#include "fused_moe_kernel_fp8.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char *err_str; \
            hipDrvGetErrorString(err, &err_str); \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << err_str << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// 封装函数
void fused_moe_kernel_fp8_constexpr_dispatcher(
    const torch::Tensor& a,           // torch.float8_e4m3fn
    const torch::Tensor& b,           // torch.float8_e4m3fn
    const torch::Tensor& c,           // torch.bfloat16
    const torch::Tensor& a_scale,     // torch.float32
    const torch::Tensor& b_scale,     // torch.float32
    const torch::Tensor& topk_weights,// torch.float32
    const torch::Tensor& sorted_token_ids, // torch.int32
    const torch::Tensor& expert_ids,  // torch.int32
    const torch::Tensor& num_tokens_post_padded, // torch.int32
    int32_t N, int32_t K, int32_t EM, int32_t num_valid_tokens,
    int32_t stride_am, int32_t stride_ak, int32_t stride_be, int32_t stride_bk,
    int32_t stride_bn, int32_t stride_cm, int32_t stride_cn,
    int32_t stride_asm, int32_t stride_ask, int32_t stride_bse,
    int32_t stride_bsk, int32_t stride_bsn,
    int64_t group_n, int64_t group_k, int64_t BLOCK_SIZE_M,
    int64_t BLOCK_SIZE_N, int64_t BLOCK_SIZE_K, int64_t GROUP_SIZE_M,
    int64_t MUL_ROUTED_WEIGHT, int64_t top_k, int64_t use_fp8_w8a8,
    int64_t use_int8_w8a16, int num_warps, int num_stages,
    unsigned int gridx, unsigned int gridy, unsigned int gridz
) {



    // 检查输入张量是否在CUDA设备上
    TORCH_CHECK(a.is_cuda(), "Input tensor 'a' must be on CUDA device");
    TORCH_CHECK(b.is_cuda(), "Input tensor 'b' must be on CUDA device");
    TORCH_CHECK(c.is_cuda(), "Input tensor 'c' must be on CUDA device");
    TORCH_CHECK(a_scale.is_cuda(), "Input tensor 'a_scale' must be on CUDA device");
    TORCH_CHECK(b_scale.is_cuda(), "Input tensor 'b_scale' must be on CUDA device");
    TORCH_CHECK(topk_weights.is_cuda(), "Input tensor 'topk_weights' must be on CUDA device");
    TORCH_CHECK(sorted_token_ids.is_cuda(), "Input tensor 'sorted_token_ids' must be on CUDA device");
    TORCH_CHECK(expert_ids.is_cuda(), "Input tensor 'expert_ids' must be on CUDA device");
    TORCH_CHECK(num_tokens_post_padded.is_cuda(), "Input tensor 'num_tokens_post_padded' must be on CUDA device");

    // 获取张量的设备指针
    hipDeviceptr_t a_ptr = reinterpret_cast<hipDeviceptr_t>(a.data_ptr());
    hipDeviceptr_t b_ptr = reinterpret_cast<hipDeviceptr_t>(b.data_ptr());
    hipDeviceptr_t c_ptr = reinterpret_cast<hipDeviceptr_t>(c.data_ptr());
    hipDeviceptr_t a_scale_ptr = reinterpret_cast<hipDeviceptr_t>(a_scale.data_ptr());
    hipDeviceptr_t b_scale_ptr = reinterpret_cast<hipDeviceptr_t>(b_scale.data_ptr());
    hipDeviceptr_t topk_weights_ptr = reinterpret_cast<hipDeviceptr_t>(topk_weights.data_ptr());
    hipDeviceptr_t sorted_token_ids_ptr = reinterpret_cast<hipDeviceptr_t>(sorted_token_ids.data_ptr());
    hipDeviceptr_t expert_ids_ptr = reinterpret_cast<hipDeviceptr_t>(expert_ids.data_ptr());
    hipDeviceptr_t num_tokens_post_padded_ptr = reinterpret_cast<hipDeviceptr_t>(num_tokens_post_padded.data_ptr());

    printf("a_ptr = %p\n", a_ptr);
    printf("b_ptr = %p\n", b_ptr);
    printf("c_ptr = %p\n", c_ptr);
    printf("a_scale_ptr = %p\n", a_scale_ptr);
    printf("b_scale_ptr = %p\n", b_scale_ptr);
    printf("topk_weights_ptr = %p\n", topk_weights_ptr);
    printf("sorted_token_ids_ptr = %p\n", sorted_token_ids_ptr);
    printf("expert_ids_ptr = %p\n", expert_ids_ptr);
    printf("num_tokens_post_padded_ptr = %p\n", num_tokens_post_padded_ptr);

    std::cout << "N = " << N << '\n';
    std::cout << "K = " << K << '\n';
    std::cout << "EM = " << EM << '\n';
    std::cout << "num_valid_tokens = " << num_valid_tokens << '\n';
    std::cout << "stride_am = " << stride_am << '\n';
    std::cout << "stride_ak = " << stride_ak << '\n';
    std::cout << "stride_be = " << stride_be << '\n';
    std::cout << "stride_bk = " << stride_bk << '\n';
    std::cout << "stride_bn = " << stride_bn << '\n';
    std::cout << "stride_cm = " << stride_cm << '\n';
    std::cout << "stride_cn = " << stride_cn << '\n';
    std::cout << "stride_asm = " << stride_asm << '\n';
    std::cout << "stride_ask = " << stride_ask << '\n';
    std::cout << "stride_bse = " << stride_bse << '\n';
    std::cout << "stride_bsk = " << stride_bsk << '\n';
    std::cout << "stride_bsn = " << stride_bsn << '\n';
    std::cout << "group_n = " << group_n << '\n';
    std::cout << "group_k = " << group_k << '\n';
    std::cout << "BLOCK_SIZE_M = " << BLOCK_SIZE_M << '\n';
    std::cout << "BLOCK_SIZE_N = " << BLOCK_SIZE_N << '\n';
    std::cout << "BLOCK_SIZE_K = " << BLOCK_SIZE_K << '\n';
    std::cout << "GROUP_SIZE_M = " << GROUP_SIZE_M << '\n';
    std::cout << "MUL_ROUTED_WEIGHT = " << MUL_ROUTED_WEIGHT << '\n';
    std::cout << "top_k = " << top_k << '\n';
    std::cout << "use_fp8_w8a8 = " << use_fp8_w8a8 << '\n';
    std::cout << "use_int8_w8a16 = " << use_int8_w8a16 << '\n';
    std::cout << "num_warps = " << num_warps << '\n';
    std::cout << "num_stages = " << num_stages << '\n';
    std::cout << "gridx = " << gridx << '\n';
    std::cout << "gridy = " << gridy << '\n';
    std::cout << "gridz = " << gridz << '\n';

    // 调用原始CUDA函数
    hipError_t result = fused_moe_kernel_fp8_constexpr_dispatcher_with_grid(
        hipStreamPerThread,
        a_ptr, b_ptr, c_ptr, a_scale_ptr, b_scale_ptr, topk_weights_ptr,
        sorted_token_ids_ptr, expert_ids_ptr, num_tokens_post_padded_ptr,
        N, K, EM, num_valid_tokens, stride_am, stride_ak, stride_be, stride_bk,
        stride_bn, stride_cm, stride_cn, stride_asm, stride_ask, stride_bse,
        stride_bsk, stride_bsn, group_n, group_k, BLOCK_SIZE_M, BLOCK_SIZE_N,
        BLOCK_SIZE_K, GROUP_SIZE_M, MUL_ROUTED_WEIGHT, top_k, use_fp8_w8a8,
        use_int8_w8a16, num_warps, num_stages, gridx, gridy, gridz
    );
    CUDA_CHECK(result);

    // 检查CUDA函数调用是否成功
    if (result != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed");
    }
}

// 绑定到Python模块
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_moe_kernel_fp8_constexpr_dispatcher", &fused_moe_kernel_fp8_constexpr_dispatcher, "Fused MoE Kernel FP8 Dispatcher");
}