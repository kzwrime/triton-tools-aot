#include "hip/hip_runtime.h"
// nvcc test_add_kernel.cu ./add_kernel.c ./kernels/*.c -lcuda

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <typeinfo>

#include "add_kernel.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char *err_str; \
            hipDrvGetErrorString(err, &err_str); \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << err_str << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

template <typename T>
__global__ void vec_add_kernel(const T* vecA, const T* vecB, T* vecC, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        vecC[idx] = vecA[idx] + vecB[idx];
    }
}

template <typename T>
void print_vector(const std::vector<T>& vec, const std::string& name, int head = 5, int tail = 5) {
    std::cout << name << " (first " << head << " and last " << tail << " elements):" << std::endl;
    for (int i = 0; i < head && i < vec.size(); ++i) {
        std::cout << vec[i] << " ";
    }
    std::cout << "... ";
    for (int i = std::max(static_cast<int>(vec.size()) - tail, 0); i < vec.size(); ++i) {
        std::cout << vec[i] << " ";
    }
    std::cout << std::endl;
}

template <typename T>
void vec_add(int n) {
    std::cout << "Initializing vectors of size " << n << " with type " << typeid(T).name() << std::endl;

    // Allocate host memory
    std::vector<T> h_vecA(n), h_vecB(n), h_vecC(n), h_vecC_ref(n);
    for (int i = 0; i < n; ++i) {
        h_vecA[i] = static_cast<T>(i);
        h_vecB[i] = static_cast<T>(i * 2);
        h_vecC_ref[i] = h_vecA[i] + h_vecB[i];  // Precompute reference result on CPU
    }

    // Print input vectors
    print_vector(h_vecA, "A");
    print_vector(h_vecB, "B");

    // Allocate device memory
    std::cout << "Allocating device memory..." << std::endl;
    hipDeviceptr_t d_vecA, d_vecB, d_vecC;
    CUDA_CHECK(hipMalloc(&d_vecA, n * sizeof(T)));
    CUDA_CHECK(hipMalloc(&d_vecB, n * sizeof(T)));
    CUDA_CHECK(hipMalloc(&d_vecC, n * sizeof(T)));

    // Copy data to device
    std::cout << "Copying data to device..." << std::endl;
    CUDA_CHECK(hipMemcpyHtoD(d_vecA, h_vecA.data(), n * sizeof(T)));
    CUDA_CHECK(hipMemcpyHtoD(d_vecB, h_vecB.data(), n * sizeof(T)));

    // Launch kernel
    std::cout << "Launching kernel..." << std::endl;
    int threadsPerBlock = 128;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    // vec_add_kernel<T><<<blocksPerGrid, threadsPerBlock>>>((const T*)d_vecA, (const T*)d_vecB, (T*)d_vecC, n);
    CUDA_CHECK(add_kernel_constexpr_dispatcher_with_grid(
        hipStreamPerThread, d_vecA, d_vecB, d_vecC, n, threadsPerBlock, 4, 3, blocksPerGrid, 1, 1)
    );

    // Check for kernel launch errors
    CUDA_CHECK(hipCtxSynchronize());
    std::cout << "Kernel execution completed." << std::endl;

    // Copy result back to host
    std::cout << "Copying result back to host..." << std::endl;
    CUDA_CHECK(hipMemcpyDtoH(h_vecC.data(), d_vecC, n * sizeof(T)));

    // Print reference and CUDA results
    print_vector(h_vecC_ref, "Reference result (A + B)");
    print_vector(h_vecC, "CUDA result (A + B)");

    // Verify the result
    std::cout << "Verifying result..." << std::endl;
    for (int i = 0; i < n; ++i) {
        if (h_vecC[i] != h_vecC_ref[i]) {
            std::cerr << "Error at index " << i << ": expected " << h_vecC_ref[i] << ", got " << h_vecC[i] << std::endl;
            std::cerr << "A[" << i << "] = " << h_vecA[i] << ", B[" << i << "] = " << h_vecB[i] << std::endl;
            break;
        }
    }
    std::cout << "Result verification completed." << std::endl;

    // Free device memory
    std::cout << "Freeing device memory..." << std::endl;
    CUDA_CHECK(hipFree(d_vecA));
    CUDA_CHECK(hipFree(d_vecB));
    CUDA_CHECK(hipFree(d_vecC));
}

int main() {
    // Initialize CUDA
    std::cout << "Initializing CUDA..." << std::endl;
    CUDA_CHECK(hipInit(0));

    // Create CUDA context
    hipDevice_t device;
    CUDA_CHECK(hipDeviceGet(&device, 0));
    hipCtx_t context;
    CUDA_CHECK(hipCtxCreate(&context, 0, device));

    std::cout << "load_vector_add..." << std::endl;
    load_add_kernel();
    std::cout << "load_vector_add finished" << std::endl;

    // Set vector length and type
    int n = 100000;
    vec_add<float>(n);  // Example with float type

    // Destroy CUDA context
    std::cout << "Destroying CUDA context..." << std::endl;
    CUDA_CHECK(hipCtxDestroy(context));

    std::cout << "Program completed successfully." << std::endl;
    return 0;
}